#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define CheckErrorUtil(err) CheckError(err, __FUNCTION__, __LINE__)
#define CheckErrorMsgUtil(err, msg) CheckErrorMsg(err, msg, __FUNCTION__, __LINE__)

inline void CheckError(hipError_t const err, char const* const fun, const int line)
{
    if (err)
    {
        printf("CUDA Error Code[%d]: %s\n%s() Line:%d\n", err, hipGetErrorString(err), fun, line);
        exit(1);
    }
}

inline void CheckErrorMsg(hipError_t const err, char const* const msg, char const* const fun, int const line)
{
    if (err)
    {
        printf("CUDA Error Code[%d]: %s\n%s() Line:%d\n%s\n", err, hipGetErrorString(err), fun, line, msg);
        exit(1);
    }
}

void GenerateTestArrays(int const N, float* const a, float* const b, float* const c, float* const ref);
void CompareArrays(int const N, float const* const a, float const* const b);

__global__ void ArraysSum(float* const a, float* const b, float* const c, int const N)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i < N)
        c[i] = a[i] + b[i];
}

void GenerateTestArrays(int const N, float* const a, float* const b, float* const c, float* const ref)
{
    int i;

    srand((unsigned)time(NULL));

    for(i = 0; i < N; i++)
    {
        a[i] = (float)rand();
        b[i] = (float)rand();
        c[i] = 0.0f;
        ref[i] = a[i] + b[i];
    }
}

void CompareArrays(int const N, float const* const a, float const* const b)
{
    int i;
    int different = 0;

    for(i = 0; i < N; i++)
    {
        different = (a[i] != b[i]);
        if(different)
            break;
    }

    if(different)
    {
        printf("Bummer.  The arrays do not match.\n");
    }
    else
    {
        printf("Awesome!  The arrays match.\n");
    }
}

int main()
{
    dim3 gridSize;
    dim3 blockSize;

    int    const N       = 40960000;
    size_t const N_BYTES = N * sizeof(float);
    int const BLOCK_SIZE = 512;

    float *aH, *bH, *cH, *refH;
    float *aD, *bD, *cD;

    aH = (float*)malloc(N_BYTES);
    bH = (float*)malloc(N_BYTES);
    cH = (float*)malloc(N_BYTES);
    refH = (float*)malloc(N_BYTES);

    printf("\n\nGenerating 2 random float arrays on Host - each of size %lu bytes...\n", N_BYTES);
    GenerateTestArrays(N, aH, bH, cH, refH);

    printf("Allocating %lu bytes on Device GPU to store the 2 generated arrays...\n", 2 * N_BYTES);
    CheckErrorUtil(hipMalloc((void**)&aD, N_BYTES));
    CheckErrorUtil(hipMalloc((void**)&bD, N_BYTES));

    printf("Allocating %lu bytes on Device GPU to store the result array after summing the 2 arrays...\n", N_BYTES);
    CheckErrorUtil(hipMalloc((void**)&cD, N_BYTES));

    printf("Copying 2 arrays from Host to Device GPU...\n");
    CheckErrorUtil(hipMemcpy(aD, aH, N_BYTES, hipMemcpyHostToDevice));
    CheckErrorUtil(hipMemcpy(bD, bH, N_BYTES, hipMemcpyHostToDevice));

    blockSize.x = BLOCK_SIZE; blockSize.y = 1; blockSize.z = 1;
    gridSize.x = ((N + BLOCK_SIZE - 1) / BLOCK_SIZE); gridSize.y = 1; gridSize.z = 1;

    printf("Summing the 2 arrays and storing the result array on Device GPU...\n");
    ArraysSum<<<gridSize, blockSize>>>(aD, bD, cD, N);

    // Get Errors from kernel
    printf("Synchronizing the Device GPU memory before copying the result array back to Host...\n");
    CheckErrorUtil(hipDeviceSynchronize());
    CheckErrorUtil(hipGetLastError());

    printf("Copying result array from Device GPU to Host...\n");
    CheckErrorUtil(hipMemcpy(cH, cD, N_BYTES, hipMemcpyDeviceToHost));

    printf("Comparing expected result array stored on Host with actual result calculated on Device GPU...\n");
    CompareArrays(N, cH, refH);

    CheckErrorUtil(hipFree(aD));
    CheckErrorUtil(hipFree(bD));
    CheckErrorUtil(hipFree(cD));

    printf("Freeing %lu bytes on Device GPU...\n", 3 * N_BYTES);
    hipFree(aD);
    hipFree(bD);
    hipFree(cD);

    printf("Freeing memory on Host...\n");
    free(aH); free(bH); free(cH); free(refH);

    CheckErrorUtil(hipDeviceReset());
    printf("Resetting Device GPU as though nothing ever happened!\n\n\n");
    hipDeviceReset();

    return 0;
}
